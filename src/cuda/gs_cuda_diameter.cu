#include "hip/hip_runtime.h"
#include <hipblas.h>
#include "gs_cuda_diameter.h"

__global__ void
diameter(int    nodes,
	 int   *degrees,
	 int   *cells,
	 int    padding,
	 float *eccentricities);

GSAPI int
gs_cuda_diameter(const matrix_t *matrix)
{
  float ecc;
  int ind;
  int *degrees_device, *data_device;
  float *ecc_device;
  dim3 block(16);
  dim3 grid(matrix->nodes / 16 + 1);

  hipMalloc((void**) &degrees_device, matrix->nodes * sizeof(int));
  hipMalloc((void**) &data_device,    matrix->size);
  hipMalloc((void**) &ecc_device,     matrix->nodes * sizeof(float));

  hipMemcpy(degrees_device, matrix->degrees, matrix->nodes * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(data_device,    matrix->cells,    matrix->size,                hipMemcpyHostToDevice);

  diameter<<<grid, block>>>(matrix->nodes, degrees_device, data_device, matrix->davg, ecc_device);
  ind = hipblasIsamax(matrix->nodes, ecc_device, 1);

  hipMemcpy(&ecc, ecc_device + ind, sizeof(float), hipMemcpyDeviceToHost);

  return ecc;
}
